/* Copyright (c) 1993-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>


// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}


#include <mma.h>
using namespace nvcuda;

// Must be multiples of 16 for wmma code to work
#define MATRIX_M 16384
#define MATRIX_N 16384
#define MATRIX_K 16384



// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;


// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16. 
//  3) Neither A nor B are transposed.
// Note: This is NOT a high performance example but is for demonstration purposes only
//       For a high performance code please use the GEMM provided in cuBLAS.
__global__ void wmma_example(half *a, half *b, float *c, int M, int N, int K, float alpha, float beta) {
   // Leading dimensions. Packed with no transpositions.
   int lda = M;
   int ldb = K;
   int ldc = M;

   // Tile using a 2D grid
   int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
   int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
 
   // Declare the fragments
   wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
   wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

   wmma::fill_fragment(acc_frag, 0.0f);

   // Loop over k
   for (int i = 0; i < K; i += WMMA_K) {
      int aRow = warpM * WMMA_M;
      int aCol = i;

      int bRow = i;
      int bCol = warpN * WMMA_N;

      // Bounds checking
      if (aRow < M && aCol < K && bRow < K && bCol < N) {
         // Load the inputs
         wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);
         wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);
 
         // Perform the matrix multiplication
         wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

      }
   }

   // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
   int cRow = warpM * WMMA_M;
   int cCol = warpN * WMMA_N;

   if (cRow < M && cCol < N) {
      wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_col_major);


      for(int i=0; i < c_frag.num_elements; i++) {
         c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
      }

      // Store the output
      wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);
   }
}


/*
   // Leading dimensions. Packed with no transpositions.
   int lda = M;
   int ldb = K;
   int ldc = M;
*/

// 32*32 256 threads
__global__ void fp16gemm(half *A, half *B, float *C, int M, int N, int K, float alpha, float beta) {

	__shared__ half sh_A[512];
	__shared__ half sh_B[512];  // 2*32*8
        
	float4 reg_C;
	half2 reg_A[2];
	half  reg_B;
	
	int im8 = threadIdx.x & 7;
	int id8 = threadIdx.x >> 3;
	int im32 = threadIdx.x & 31;
	int id32 = threadIdx.x >> 5;

	// Compute block's starting coordinate
	int block_base_x = blockIdx.y << 5;
	int block_base_y = blockIdx.x << 5;

	//Load C from global memory to register file
	float4 *C_start = (float4 *) (C + block_base_x*M + block_base_y + (im8<<2) + (id8)*M);
    reg_C = *C_start;

	//load B from global memory to shared memory
	half *A_start = (A + block_base_y + (im32) + (id32)*M); 
	*(sh_A + threadIdx.x) = *(A_start);

	//load A from global memory to shared memory
	half *B_start = (B + K*block_base_x + (id32) + (im32)*K); 
	*(sh_B + threadIdx.x) = *(B_start);

	int double_buffer = 0;
#pragma unroll
	for(int k=0; k<K; k+=8)
	{
		__syncthreads();
		int A_offset = double_buffer + (im8 << 2);
		int B_offset = double_buffer + id8;
			
#pragma unroll
		for (int i=0; i<8; ++i)	
		{
			reg_A[0] = *((half2*) (sh_A + A_offset)); 
			reg_B = sh_B[B_offset]; 

			reg_C.x += (float)__hmul(reinterpret_cast<half&>(reg_A[0].x), reg_B);
			reg_C.y += (float)__hmul(reinterpret_cast<half&>(reg_A[0].y), reg_B);
			
			//reg_C.y = hfma(reg_A[0].y, reg_B, reg_C.y);
			
			reg_A[1] = *(half2*) (sh_A + A_offset + 2); 
			reg_C.z += (float)__hmul(reinterpret_cast<half&>(reg_A[1].x), reg_B);
			reg_C.w += (float)__hmul(reinterpret_cast<half&>(reg_A[1].y), reg_B);
			//reg_C[1] = __hfma2(reg_A[1], reg_B, reg_C[1]);
			
			A_offset += 32;
			B_offset += 32;
		}

		double_buffer ^= 256;

		if (k+8 < K){
			A_start += M << 3; 
			*(sh_A + double_buffer + threadIdx.x) = *(A_start);

			B_start += 8; 
			*(sh_B + double_buffer + threadIdx.x) = *(B_start);
		}
	}
	
	*(C_start) = reg_C;
	
	//*(C_start) = __half22float2(reg_C[0]);
	//*(C_start+1) = __half22float2(reg_C[1]);
}

__global__ void convertFp32ToFp16 (half *out, float *in, int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      out[idx] = in[idx];
   }
}

int main(int argc, char* argv[]) {
   float *a_fp32;
   float *b_fp32;
   half *a_fp16;
   half *b_fp16;

   float *c;
   float *c_cublas;
   float *c_wmma;

   float *c_host_cublas;
   float *c_host_wmma;
   
   hiprandGenerator_t gen;
   hipblasHandle_t cublasHandle;
   
   hipEvent_t startWMMA;
   hipEvent_t stopWMMA;
   
   hipEvent_t startcublas;
   hipEvent_t stopcublas;
   
   cudaErrCheck(hipEventCreate(&startWMMA));
   cudaErrCheck(hipEventCreate(&stopWMMA));
   
   cudaErrCheck(hipEventCreate(&startcublas));
   cudaErrCheck(hipEventCreate(&stopcublas));
   
   
   cublasErrCheck(hipblasCreate(&cublasHandle));
   
   // Use tensor cores
   cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));
   
   cudaErrCheck(hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
   cudaErrCheck(hipMalloc((void**)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));

   cudaErrCheck(hipMalloc((void**)&c, MATRIX_M * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&c_cublas, MATRIX_M * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&c_wmma, MATRIX_M * MATRIX_N * sizeof(float)));

   c_host_cublas = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
   c_host_wmma = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

   curandErrCheck(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
   curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL));

   curandErrCheck(hiprandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K));
   curandErrCheck(hiprandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N));

   // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (a_fp16, a_fp32, MATRIX_M * MATRIX_K);
   convertFp32ToFp16 <<< (MATRIX_K * MATRIX_N + 255) / 256, 256 >>> (b_fp16, b_fp32, MATRIX_K * MATRIX_N);

   curandErrCheck(hiprandGenerateUniform(gen, c, MATRIX_M * MATRIX_N));
   
   curandErrCheck(hiprandDestroyGenerator(gen));
   
   cudaErrCheck(hipMemcpy(c_cublas, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));
   cudaErrCheck(hipMemcpy(c_wmma, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));

   float alpha = 1.0f;
   float beta = 0.0f;


   printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
   
   // First: using WMMA
   dim3 gridDim;
   dim3 blockDim;
 
   // blockDim.x must be a multple of warpSize
   // 128x4 means we have 16 warps and a block computes a 64x64 output tile
   blockDim.x = 128;
   blockDim.y = 4;
   
   dim3 gridDim2;
   dim3 blockDim2;
   gridDim2.x = MATRIX_M/32; gridDim2.y = MATRIX_N/32;  gridDim2.z = 1;
   blockDim2.x = 256; blockDim2.y = 1; blockDim2.z = 1; 

   gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
   gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);
   
   printf("Running with wmma...\n");
   cudaErrCheck(hipEventRecord(startWMMA));
   //wmma_example <<< gridDim, blockDim >>> (a_fp16, b_fp16, c_wmma, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
   fp16gemm <<< gridDim2, blockDim2 >>> (a_fp16, b_fp16, c_wmma, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
   cudaErrCheck(hipEventRecord(stopWMMA));
   
   // Now using cuBLAS
   printf("Running with cuBLAS...\n");
   cudaErrCheck(hipEventRecord(startcublas));
   cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                MATRIX_M, MATRIX_N, MATRIX_K, 
                &alpha,
                a_fp16, HIP_R_16F, MATRIX_M,
                b_fp16, HIP_R_16F, MATRIX_K,
                &beta, 
                c_cublas, HIP_R_32F, MATRIX_M,
                HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
   cudaErrCheck(hipEventRecord(stopcublas));

   // Error checking
   printf("\nChecking results...\n");
   cudaErrCheck(hipMemcpy(c_host_wmma, c_wmma, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
   cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
   
   // 0.01% relative tolerance. 1e-5 absolute tolerance.
   int errors = 0;
   for (int i = 0; i < MATRIX_M * MATRIX_N; i++) {
      float v1 = c_host_wmma[i];
      float v2 = c_host_cublas[i];
      if (v1 / v2 > 1.0001 || v2 / v1 > 1.0001 || abs(v1 - v2) > 1e-5) {
         errors++;
         if (errors < 10) printf("%f %f\n", v1, v2);
      }
   }
   
   if (errors > 0) {
      printf("WMMA does not agree with cuBLAS! %d errors!\n", errors);
   }
   else {
      printf("Results verified: cublas and WMMA agree.\n\n");
      float wmmaTime;
      float cublasTime;
      cudaErrCheck(hipEventSynchronize(stopWMMA));
      cudaErrCheck(hipEventSynchronize(stopcublas));
      cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWMMA, stopWMMA));
      cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
      printf("wmma took %fms\n", wmmaTime);
      printf("cublas took %fms\n", cublasTime);

      printf("\nFor a faster code using wmma you should check out the cudaTensorCoreGemm sample in the CUDA Toolkit.\nThis code was written as a demo only!\n\n");
   }
   
   
   cudaErrCheck(hipEventDestroy(startWMMA));
   cudaErrCheck(hipEventDestroy(stopWMMA));

   cudaErrCheck(hipEventDestroy(startcublas));             
   cudaErrCheck(hipEventDestroy(stopcublas));
   
   cudaErrCheck(hipFree(a_fp32));
   cudaErrCheck(hipFree(b_fp32));
   cudaErrCheck(hipFree(a_fp16));
   cudaErrCheck(hipFree(b_fp16));

   cudaErrCheck(hipFree(c));
   cudaErrCheck(hipFree(c_cublas));
   cudaErrCheck(hipFree(c_wmma));
   
   free(c_host_cublas);
   free(c_host_wmma);

   cudaErrCheck(hipDeviceReset());
   return 0;
}
