#include "hip/hip_runtime.h"
 
 //  nvcc -o gemm -arch=sm_70 -lcublas -lcurand fp16.cu

#include <stdio.h>
#include <hiprand.h>
#include <hipblas.h>

#include <hip/hip_fp16.h>


// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}


#include <mma.h>
using namespace nvcuda;

// Must be multiples of 16 for wmma code to work  16384
#define MATRIX_M 49
#define MATRIX_N 48
#define MATRIX_K 48



// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;


__global__ void convertFp32ToFp16 (half *out, float *in, int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      out[idx] = in[idx];
   }
}

// 64 thread
__global__ void fp16gemm_16x16(float *A, float *B, float *C, int M, int N, int K, float alpha, float beta) {

	__shared__ half sh_A[256];
	__shared__ half sh_B[256];  // 2*16*8
        
	float2 reg_C[2];
	half2 reg_A[4];
	half2  reg_B[2];
	
	reg_C[0].x = 0.f;
	reg_C[0].y = 0.f;
	reg_C[1].x = 0.f;
	reg_C[1].y = 0.f;
	
	int im4 = threadIdx.x & 3;
	int id4 = threadIdx.x >> 2;
	int im8 = threadIdx.x & 7;
	int id8 = threadIdx.x >> 3;
	int im16 = threadIdx.x & 15;
	int id16 = threadIdx.x >> 4;
	
	int thread2 = threadIdx.x << 1;

    // Compute block's starting coordinate
    int block_base_x = blockIdx.y << 4;
    int block_base_y = blockIdx.x << 4;

    //load A from global memory to shared memory  sgemm中A和B是分别用两个warp载入的
    float2 *A_start = (float2*) (A + block_base_y + (im8 << 1) + (id8)*M);
    *((half2*)(sh_A + thread2)) = __float22half2_rn(*(A_start));

    //load B from global memory to shared memory
    float2 *B_start = (float2*) (B + K*(im16+block_base_x) + (id16 << 1));
    *((half2*) (sh_B + thread2)) = __float22half2_rn(*(B_start));

    int double_buffer = 0;
#pragma unroll
    for(int k = 0; k < K; k += 8)
	{
        __syncthreads();
        int A_offset = double_buffer + (im4 << 2);
        int B_offset = double_buffer + (id4 << 1);	
		
#pragma unroll
        for (int i=0; i<8; i+=2)   // 全部展开有register spill吗
		{
			reg_A[0] = *((half2*)(sh_A + A_offset));
			reg_A[1] = *((half2*)(sh_A + A_offset + 2));
			reg_A[2] = *((half2*)(sh_A + A_offset + 16));
			reg_A[3] = *((half2*)(sh_A + A_offset + 18));
            //reg_A[0] = sh_A[A_offset];    // A_offset+0 ~ A_offset+3 为什么不用向量呢
			reg_B[0] = *((half2*)(sh_B + B_offset));
			reg_B[1].x = reg_B[0].y;
			reg_B[0].y = reg_B[0].x;
			reg_B[1].y = reg_B[1].x;
            //reg_A[1] = sh_A[A_offset+1];  // 为了避免bank冲突, 这8个寄存器都不是连续的【4个就不会重复】，因此不能使用向量传输指令
            //reg_A[2] = sh_A[A_offset+2];
            //reg_A[3] = sh_A[A_offset+3];
			
            //reg_C.x = fma(reg_A[0], reg_B[0], reg_C.x); // reg_C.x = reg_A[0]*reg_B[0] + reg_A[4]*reg_B[1]
            //reg_C.y = fma(reg_A[1], reg_B[0], reg_C.y);
			reg_C[0] = __half22float2(__hfma2(reg_A[0], reg_B[0], __float22half2_rn(reg_C[0])));
            //reg_C.z = fma(reg_A[2], reg_B[0], reg_C.z);
            //reg_C.w = fma(reg_A[3], reg_B[0], reg_C.w);
			reg_C[1] = __half22float2(__hfma2(reg_A[1], reg_B[0], __float22half2_rn(reg_C[1])));
			
			//*((float4*) (reg_A + 4)) = *((float4*)(sh_A + A_offset + 16));
			//reg_B[1] = sh_B[B_offset+1];
			//reg_A[4] = sh_A[A_offset+16];
            //reg_A[5] = sh_A[A_offset+17];
            //reg_A[6] = sh_A[A_offset+18];
            //reg_A[7] = sh_A[A_offset+19];
			A_offset += 32;
			
            //reg_C.x = fma(reg_A[4], reg_B[1], reg_C.x);
            //reg_C.y = fma(reg_A[5], reg_B[1], reg_C.y);
            //reg_C.z = fma(reg_A[6], reg_B[1], reg_C.z);
            //reg_C.w = fma(reg_A[7], reg_B[1], reg_C.w);
			reg_C[0] = __half22float2(__hfma2(reg_A[2], reg_B[1], __float22half2_rn(reg_C[0]))) ;
			reg_C[1] = __half22float2(__hfma2(reg_A[3], reg_B[1], __float22half2_rn(reg_C[1]))) ;

            B_offset += 32;
        }
		
        double_buffer ^= 128;  // 16*8
		
        if (k+8 < K)
		{
            A_start += M << 2; // half2 --> 8M
            *((half2*) (sh_A + double_buffer + thread2)) = __float22half2_rn(*(A_start));
            B_start += 4;
            *((half2*) (sh_B + double_buffer + thread2)) = __float22half2_rn(*(B_start));
        }
    }

	int ind = block_base_y + (im4<<2);     // 横、纵坐标  M=HW， K = C， N = K
	// blockIdx.x*16 < (M + (0)*16) ;  M%16 == 0 && P%2 == 0 ;
	int PQ = M;
    int C_offset = ind/(PQ)*(PQ*N) + ind%(PQ) + (id4 + block_base_x)*(PQ);
    *((float2*)(C + C_offset)) = reg_C[0];
    //C[C_offset+1] = reg_C.y;
	*((float2*)(C + C_offset + 2)) = reg_C[1];
}


// 64 thread
__global__ void fp16gemm_16x16_tensor(float *A, float *B, float *C, int M, int N, int K, float alpha, float beta) {

	__shared__ half sh_A[512];
	__shared__ half sh_B[512];  // 2*16*16
	
	int im4 = threadIdx.x & 3;
	int id4 = threadIdx.x >> 2;
	//int im8 = threadIdx.x & 7;
	//int id8 = threadIdx.x >> 3;
	//int im16 = threadIdx.x & 15;
	//int id16 = threadIdx.x >> 4;
	
	//int thread2 = threadIdx.x << 1;
	int thread4 = threadIdx.x << 2;

    // Compute block's starting coordinate
    int block_base_x = blockIdx.y << 4;
    int block_base_y = blockIdx.x << 4;

    //load A from global memory to shared memory  sgemm中A和B是分别用两个warp载入的
    float2 *A_start = (float2*) (A + block_base_y + (im4 << 2) + (id4)*M);
    *((half2*)(sh_A + thread4)) = __float22half2_rn(*(A_start));
	*((half2*)(sh_A + thread4 + 2)) = __float22half2_rn(*(A_start + 1));

    //load B from global memory to shared memory
	float2 *B_start = (float2*) (B + K*block_base_x + (im4 << 2) + (id4)*K);
    //float2 *B_start = (float2*) (B + K*(im16+block_base_x) + (id16 << 1));
    *((half2*) (sh_B + thread4)) = __float22half2_rn(*(B_start));
	*((half2*) (sh_B + thread4 + 2)) = __float22half2_rn(*(B_start + 1));

    int double_buffer = 0;
	
   // Declare the fragments
   wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
   wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
   //wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

   wmma::fill_fragment(acc_frag, 0.0f);
   
#pragma unroll
    for(int k = 0; k < K; k += 16)
	{
        __syncthreads();
		
		// Load the inputs
        wmma::load_matrix_sync(a_frag, sh_A + double_buffer, 16);
        wmma::load_matrix_sync(b_frag, sh_B + double_buffer, 16);
 
        // Perform the matrix multiplication
        wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);		

        double_buffer ^= 256;  // 16*16
		
        if (k + 16 < K)
		{
            A_start += M << 3; // half2 --> 8M
            *((half2*) (sh_A + double_buffer + thread4)) = __float22half2_rn(*(A_start));
			*((half2*) (sh_A + double_buffer + thread4 + 2)) = __float22half2_rn(*(A_start+1));
			
            B_start += 8;
            *((half2*) (sh_B + double_buffer + thread4)) = __float22half2_rn(*(B_start));
			*((half2*) (sh_B + double_buffer + thread4 + 2)) = __float22half2_rn(*(B_start+1));
        }
    }

	//int ind = block_base_y + (im4<<2);     // 横、纵坐标  M=HW， K = C， N = K
	//int PQ = M;
    //int C_offset = ind/(PQ)*(PQ*N) + ind%(PQ) + (id4 + block_base_x)*(PQ);
    //*((float2*)(C + C_offset)) = reg_C[0];
	//*((float2*)(C + C_offset + 2)) = reg_C[1];
	
	// Store the output
    wmma::store_matrix_sync(C + block_base_y + (block_base_x * M), acc_frag, M, wmma::mem_col_major);	
}


__global__ void gemm_64_16x16_3_tensor(int M, int N, int K, float *A, float *B, float *C){

   __shared__ half sh_A[512];
   __shared__ half sh_B[512];
   __shared__ float sh_C[256];

   float reg_C[4];
   reg_C[0] = 0.f;
   reg_C[1] = 0.f;
   reg_C[2] = 0.f;
   reg_C[3] = 0.f;

   int im4 = threadIdx.x & 3;
   int id4 = threadIdx.x >> 2;
   int thread4 = threadIdx.x << 2;
   
   // Compute block's starting coordinate
   int block_base_x = blockIdx.y*16;
   int block_base_y = blockIdx.x*16;


    //load A from global memory to shared memory  sgemm中A和B是分别用两个warp载入的
    float2 *A_start = (float2*) (A + block_base_y + (im4 << 2) + (id4)*M);
	if (block_base_y == 3)
	{
		if (id4 == 0)
			*((half2*)(sh_A + thread4)) = __float22half2_rn({(*(A_start)).x, 0.f});
		*((half2*)(sh_A + thread4 + 2)) = __float22half2_rn({0.f, 0.f});
	}
	else
	{
		*((half2*)(sh_A + thread4)) = __float22half2_rn(*(A_start));
		*((half2*)(sh_A + thread4 + 2)) = __float22half2_rn(*(A_start + 1));		
	}

    //load B from global memory to shared memory
	float2 *B_start = (float2*) (B + K*block_base_x + (im4 << 2) + (id4)*K);
    //float2 *B_start = (float2*) (B + K*(im16+block_base_x) + (id16 << 1));
    *((half2*) (sh_B + thread4)) = __float22half2_rn(*(B_start));
	*((half2*) (sh_B + thread4 + 2)) = __float22half2_rn(*(B_start + 1));

   int double_buffer = 0;
   
   // Declare the fragments
   wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
   wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
   //wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

   wmma::fill_fragment(acc_frag, 0.0f);
   
#pragma unroll
   for(int k=0; k<K; k+=16)
   {
       __syncthreads();
	   
		// Load the inputs
        wmma::load_matrix_sync(a_frag, sh_A + double_buffer, 16);
        wmma::load_matrix_sync(b_frag, sh_B + double_buffer, 16);
 
        // Perform the matrix multiplication
        wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);		   

       double_buffer ^= 256;

       if (k+16 < K)
	   {
           A_start += M<<3;
			if (block_base_y == 3)
			{
				if (id4 == 0)
					*((half2*)(sh_A + double_buffer  + thread4)) = __float22half2_rn({(*(A_start)).x, 0.f});
				*((half2*)(sh_A + double_buffer  + thread4 + 2)) = __float22half2_rn({0.f, 0.f});
			}
			else
			{
				*((half2*)(sh_A + double_buffer  + thread4)) = __float22half2_rn(*(A_start));
				*((half2*)(sh_A + double_buffer  + thread4 + 2)) = __float22half2_rn(*(A_start + 1));		
			}
			
           B_start += 8;
			*((half2*) (sh_B + double_buffer  + thread4)) = __float22half2_rn(*(B_start));
			*((half2*) (sh_B + double_buffer  + thread4 + 2)) = __float22half2_rn(*(B_start + 1));
       }
   }

	int ind = blockIdx.x*16 + (threadIdx.x%4)*4;
	int PQ = M; 
    int C_offset = ind/(PQ)*(PQ*N) + ind%(PQ) + (threadIdx.x/4)*(PQ) + blockIdx.y*16*(PQ);

   if (blockIdx.x < M/16)
   {
		// Store the output
		wmma::store_matrix_sync(C + block_base_y + (block_base_x * M), acc_frag, M, wmma::mem_col_major);		
   }
   else
   {
		// Store the output
		wmma::store_matrix_sync(sh_C, acc_frag, 16, wmma::mem_col_major);
		
	   reg_C[0] = sh_C[threadIdx.x*4];
	   reg_C[1] = sh_C[threadIdx.x*4 + 1];
	   reg_C[2] = sh_C[threadIdx.x*4 + 2];
	   reg_C[3] = sh_C[threadIdx.x*4 + 3];
	
       int ruler = (threadIdx.x%4)*4;
       int rag = M%16;
	   
       if (ruler < rag)
	   {
           C[C_offset] = reg_C[0];
	    }
		
       if ((ruler+1) < rag){
     	   C_offset = (ind+1)/(PQ)*(PQ*N) + (ind+1)%(PQ) + (threadIdx.x/4)*(PQ) + blockIdx.y*16*(PQ);
           C[C_offset] = reg_C[1];
		}
		
       if ((ruler+2) < rag){
			C_offset = (ind+2)/(PQ)*(PQ*N) + (ind+2)%(PQ) + (threadIdx.x/4)*(PQ) + blockIdx.y*16*(PQ);
            C[C_offset] = reg_C[2];
		}
		
       if ((ruler+3) < rag){
     	   C_offset = (ind+3)/(PQ)*(PQ*N) + (ind+3)%(PQ) + (threadIdx.x/4)*(PQ) + blockIdx.y*16*(PQ);
           C[C_offset] = reg_C[3];
		}
   }
}


__global__ void gemm_64_16x16_3(int M, int N, int K, float *A, float *B, float *C){

   __shared__ float sh_A[256];
   __shared__ float sh_B[256];
   //float* sh_B = sh + 2*16*8;

    float4 reg_C;
	reg_C.x =0.f;
	reg_C.y =0.f;
	reg_C.z =0.f;
	reg_C.w =0.f;

    float reg_A[8];
    float reg_B[2];

    // Compute block's starting coordinate
    int block_base_x = blockIdx.y*16;
    int block_base_y = blockIdx.x*16;

    //load A from global memory to shared memory  sgemm中A和B是分别用两个warp载入的
    int aoffset = block_base_y + (threadIdx.x%8)*2 + (threadIdx.x/8)*M;
    sh_A[2*threadIdx.x] = A[aoffset%(M*K)];
	sh_A[2*threadIdx.x+1] = A[(aoffset+1)%(M*K)];

    //load B from global memory to shared memory
    int boffset = K*block_base_x + (threadIdx.x/16)*2 + (threadIdx.x%16)*K;
    sh_B[2*threadIdx.x] = B[boffset%(N*K)];
	sh_B[2*threadIdx.x+1] = B[(boffset+1)%(N*K)];

    int double_buffer = 0;
#pragma unroll
    for(int k = 0; k < K; k += 8)
	{
        __syncthreads();
        int A_offset = double_buffer + (threadIdx.x%4)*4;
        int B_offset = double_buffer + ((threadIdx.x/4)*2);	
		
#pragma unroll
        for (int i=0; i<8; i+=2)   // 全部展开有register spill吗
		{
            reg_A[0] = sh_A[A_offset];    // A_offset+0 ~ A_offset+3 为什么不用向量呢
            reg_A[1] = sh_A[A_offset+1];  // 为了避免bank冲突, 这8个寄存器都不是连续的【4个就不会重复】，因此不能使用向量传输指令
            reg_A[2] = sh_A[A_offset+2];
            reg_A[3] = sh_A[A_offset+3];
			reg_A[4] = sh_A[A_offset+16];
            reg_A[5] = sh_A[A_offset+17];
            reg_A[6] = sh_A[A_offset+18];
            reg_A[7] = sh_A[A_offset+19];
			
			reg_B[0] = sh_B[B_offset];
			
            reg_C.x = fma(reg_A[0], reg_B[0], reg_C.x); // reg_C.x = reg_A[0]*reg_B[0] + reg_A[4]*reg_B[1]
            reg_C.y = fma(reg_A[1], reg_B[0], reg_C.y);
            reg_C.z = fma(reg_A[2], reg_B[0], reg_C.z);
            reg_C.w = fma(reg_A[3], reg_B[0], reg_C.w);
			
			reg_B[1] = sh_B[B_offset+1];
			
            reg_C.x = fma(reg_A[4], reg_B[1], reg_C.x);
            reg_C.y = fma(reg_A[5], reg_B[1], reg_C.y);
            reg_C.z = fma(reg_A[6], reg_B[1], reg_C.z);
            reg_C.w = fma(reg_A[7], reg_B[1], reg_C.w);

            A_offset += 32;
            B_offset += 32;
        }
		
        double_buffer ^= 128;
		
        if (k + 8 < K)
		{
            aoffset += 8*M; // float2 --> 8M
            //*((float2*) (sh_A + double_buffer + 2*threadIdx.x)) = *(A_start);
            boffset += 8;
            //*((float2*) (sh_B + double_buffer + 2*threadIdx.x)) = *(B_start);
			
			//int aoffset = block_base_y + (threadIdx.x%8)*2 + (threadIdx.x/8)*M;
			sh_A[double_buffer+2*threadIdx.x] = A[aoffset%(M*K)];
			sh_A[double_buffer+2*threadIdx.x+1] = A[(aoffset+1)%(M*K)];

			//load B from global memory to shared memory
			//int boffset = K*block_base_x + (threadIdx.x/16)*2 + (threadIdx.x%16)*K;
			sh_B[double_buffer+2*threadIdx.x] = B[boffset%(N*K)];
			sh_B[double_buffer+2*threadIdx.x+1] = B[(boffset+1)%(N*K)];
        }
    }

	int ind = blockIdx.x*16 + (threadIdx.x%4)*4;  // 横、纵坐标  M=HW， K = C， N = K
	// blockIdx.x*16 < (M + (0)*16) ;  M%16 == 0 && P%2 == 0 ;   relu = max(0, x)
    int C_offset = ind/(P*Q)*(P*Q*N) + ind%(P*Q) + (threadIdx.x/4)*(P*Q) + blockIdx.y*16*(P*Q);
    C[C_offset] = reg_C.x > 0 ? reg_C.x : 0;
    C[C_offset+1] = reg_C.y;
    C[C_offset+2] = reg_C.z;
    C[C_offset+3] = reg_C.w;
}

int main(int argc, char* argv[]) 
{
   float *a_fp32;
   float *b_fp32;
   half *a_fp16;
   half *b_fp16;

   float *c;
   float *c_cublas;
   float *c_wmma;

   float *c_host_cublas;
   float *c_host_wmma;
   
   hiprandGenerator_t gen;
   hipblasHandle_t cublasHandle;
   
   hipEvent_t startWMMA;
   hipEvent_t stopWMMA;
   
   hipEvent_t startcublas;
   hipEvent_t stopcublas;
   
   cudaErrCheck(hipEventCreate(&startWMMA));
   cudaErrCheck(hipEventCreate(&stopWMMA));
   
   cudaErrCheck(hipEventCreate(&startcublas));
   cudaErrCheck(hipEventCreate(&stopcublas));
   
   
   cublasErrCheck(hipblasCreate(&cublasHandle));
   
   // Use tensor cores
   cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));
   
   cudaErrCheck(hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
   cudaErrCheck(hipMalloc((void**)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));

   cudaErrCheck(hipMalloc((void**)&c, MATRIX_M * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&c_cublas, MATRIX_M * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&c_wmma, MATRIX_M * MATRIX_N * sizeof(float)));

   c_host_cublas = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
   c_host_wmma = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

   curandErrCheck(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
   curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL));

   curandErrCheck(hiprandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K));
   curandErrCheck(hiprandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N));

   // hiprand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (a_fp16, a_fp32, MATRIX_M * MATRIX_K);
   convertFp32ToFp16 <<< (MATRIX_K * MATRIX_N + 255) / 256, 256 >>> (b_fp16, b_fp32, MATRIX_K * MATRIX_N);

   curandErrCheck(hiprandGenerateUniform(gen, c, MATRIX_M * MATRIX_N));
   
   curandErrCheck(hiprandDestroyGenerator(gen));
   
   cudaErrCheck(hipMemcpy(c_cublas, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));
   cudaErrCheck(hipMemcpy(c_wmma, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));

   float alpha = 1.0f;
   float beta = 0.0f;


   printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
   
   // First: using WMMA
   dim3 gridDim;
   dim3 blockDim;
 
   // blockDim.x must be a multple of warpSize
   // 128x4 means we have 16 warps and a block computes a 64x64 output tile
   blockDim.x = 128;
   blockDim.y = 4;
   
   dim3 gridDim2;
   dim3 blockDim2;
   gridDim2.x = MATRIX_M/128; gridDim2.y = MATRIX_N/64;  gridDim2.z = 1;
   blockDim2.x = 256; blockDim2.y = 1; blockDim2.z = 1; 

   gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
   gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

   dim3 gridDim3;
   dim3 blockDim3;
   gridDim3.x = MATRIX_M/16; gridDim3.y = MATRIX_N/16; gridDim3.z = 1;
   blockDim3.x = 64; blockDim3.y = 1; blockDim3.z = 1;
   
   printf("Running with wmma...\n");
   cudaErrCheck(hipEventRecord(startWMMA));
   
   //convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (a_fp16, a_fp32, MATRIX_M * MATRIX_K);
   //convertFp32ToFp16 <<< (MATRIX_K * MATRIX_N + 255) / 256, 256 >>> (b_fp16, b_fp32, MATRIX_K * MATRIX_N);
   //fp16gemm_16x16<<< gridDim3, blockDim3 >>>(a_fp32, b_fp32, c_wmma, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
   //fp16gemm_16x16_tensor<<< gridDim3, blockDim3 >>>(a_fp32, b_fp32, c_wmma, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
   //gemm_64_16x16_3_tensor<<< gridDim3, blockDim3 >>>(MATRIX_M, MATRIX_N, MATRIX_K, a_fp32, b_fp32, c_wmma);
   gemm_64_16x16_3 <<< gridDim3, blockDim3 >>>(MATRIX_M, MATRIX_N, MATRIX_K, a_fp32, b_fp32, c_wmma);
   cudaErrCheck(hipEventRecord(stopWMMA));
   
   // Now using cuBLAS
   printf("Running with cuBLAS...\n");
   cudaErrCheck(hipEventRecord(startcublas));
   cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                MATRIX_M, MATRIX_N, MATRIX_K, 
                &alpha,
                a_fp16, HIP_R_16F, MATRIX_M,
                b_fp16, HIP_R_16F, MATRIX_K,
                &beta, 
                c_cublas, HIP_R_32F, MATRIX_M,
                HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
   cudaErrCheck(hipEventRecord(stopcublas));

   // Error checking
   printf("\nChecking results...\n");
   cudaErrCheck(hipMemcpy(c_host_wmma, c_wmma, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
   cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
   
   // 0.01% relative tolerance. 1e-5 absolute tolerance.
   int errors = 0;
   for (int i = 0; i < MATRIX_M * MATRIX_N; i++) {
      float v1 = c_host_wmma[i];
      float v2 = c_host_cublas[i];
      if (v1 / v2 > 1.0001 || v2 / v1 > 1.0001 || abs(v1 - v2) > 1e-5) {
         errors++;
         if (errors < 10) printf("%f %f\n", v1, v2);
      }
   }
   
   //if (errors > 0) {
   //   printf("WMMA does not agree with cuBLAS! %d errors!\n", errors);
   //}
   //else {
      printf("Results verified: cublas and WMMA agree.\n\n");
      float wmmaTime;
      float cublasTime;
      cudaErrCheck(hipEventSynchronize(stopWMMA));
      cudaErrCheck(hipEventSynchronize(stopcublas));
      cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWMMA, stopWMMA));
      cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
      printf("wmma took %fms\n", wmmaTime);
      printf("cublas took %fms\n", cublasTime);

      printf("\nFor a faster code using wmma you should check out the cudaTensorCoreGemm sample in the CUDA Toolkit.\nThis code was written as a demo only!\n\n");
   //}
   
   
   cudaErrCheck(hipEventDestroy(startWMMA));
   cudaErrCheck(hipEventDestroy(stopWMMA));

   cudaErrCheck(hipEventDestroy(startcublas));             
   cudaErrCheck(hipEventDestroy(stopcublas));
   
   cudaErrCheck(hipFree(a_fp32));
   cudaErrCheck(hipFree(b_fp32));
   cudaErrCheck(hipFree(a_fp16));
   cudaErrCheck(hipFree(b_fp16));

   cudaErrCheck(hipFree(c));
   cudaErrCheck(hipFree(c_cublas));
   cudaErrCheck(hipFree(c_wmma));
   
   free(c_host_cublas);
   free(c_host_wmma);

   cudaErrCheck(hipDeviceReset());
   return 0;
}
